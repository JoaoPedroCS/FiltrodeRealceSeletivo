#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

typedef struct {
    unsigned char r, g, b;
} Pixel;

void Allocate_memmory(Pixel **d_original_p, unsigned char **d_grayscale_p, unsigned char **d_blurred_p,
                      Pixel **d_final_p, int width, int height) {
    int n = width * height;
    hipMalloc((void **)d_original_p, n * sizeof(Pixel));
    hipMalloc((void **)d_grayscale_p, n * sizeof(unsigned char));
    hipMalloc((void **)d_blurred_p, n * sizeof(unsigned char));
    hipMalloc((void **)d_final_p, n * sizeof(Pixel));
}

void Free_vectors(Pixel **d_original_p, unsigned char **d_grayscale_p, unsigned char **d_blurred_p,
                  Pixel **d_final_p) {
    hipFree(*d_original_p);
    hipFree(*d_grayscale_p);
    hipFree(*d_blurred_p);
    hipFree(*d_final_p);
}

__global__ void gray_scale_transformation(const Pixel *orig, unsigned char *gray, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= n) return;
    unsigned char r = orig[idx].r;
    unsigned char g = orig[idx].g;
    unsigned char b = orig[idx].b;

    gray[idx] = (unsigned char)(0.299f * r + 0.587f * g + 0.114f * b);
}

__global__ void blur_transformation(const Pixel *orig, const unsigned char *gray,
                                    unsigned char *blur, int width, int height, int M) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int n = width * height;
    if (idx >= n) return;

    Pixel p = orig[idx];
    int radius = ((p.r + p.g + p.b) % M) + 1;
    int x = idx % width;
    int y = idx / width;

    int x0 = max(0, x - radius);
    int x1 = min(width - 1, x + radius);
    int y0 = max(0, y - radius);
    int y1 = min(height - 1, y + radius);

    unsigned int sum = 0;
    int count = 0;
    for (int yy = y0; yy <= y1; ++yy) {
        int base = yy * width;
        for (int xx = x0; xx <= x1; ++xx) {
            sum += gray[base + xx];
            ++count;
        }
    }
    blur[idx] = (unsigned char)(sum / (count ? count : 1));
}

__device__ unsigned char clamp(int v) {
    if (v < 0) return 0;
    if (v > 255) return 255;
    return (unsigned char)v;
}

__global__ void sharpen_kernel(const Pixel *orig, const unsigned char *blur,
                               Pixel *out, int n, int limiar, float sharpen_factor) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = idx; i < n; i += stride) {
        Pixel p = orig[i];
        unsigned char b = blur[i];

        if (p.r > limiar) {
            float new_r = (float)p.r + sharpen_factor * ((float)p.r - (float)b);
            float new_g = (float)p.g + sharpen_factor * ((float)p.g - (float)b);
            float new_b = (float)p.b + sharpen_factor * ((float)p.b - (float)b);

            out[i].r = clamp((int)new_r);
            out[i].g = clamp((int)new_g);
            out[i].b = clamp((int)new_b);
        } else {
            out[i] = p;
        }
    }
}

int main(int argc, char *argv[]) {
    if (argc != 6) {
        fprintf(stderr, "Erro: Número incorreto de argumentos.\n");
        fprintf(stderr, "Uso: %s <input.ppm> <output.ppm> <M> <limiar> <sharpen_factor>\n", argv[0]);
        fprintf(stderr, "Onde:\n");
        fprintf(stderr, "  M:               Inteiro >= 1 para a fórmula do raio.\n");
        fprintf(stderr, "  limiar:          Inteiro [0-255] para o critério de sharpen.\n");
        fprintf(stderr, "  sharpen_factor:  Float para a intensidade do sharpen (ex: 1.2).\n");
        return 1;
    }

    const char *input_filename = argv[1];
    const char *output_filename = argv[2];
    int M = atoi(argv[3]);
    int limiar = atoi(argv[4]);
    float sharpen_factor = atof(argv[5]);

    if (M < 1) {
        fprintf(stderr, "Erro: M deve ser um inteiro maior ou igual a 1.\n");
        return 1;
    }
    if (limiar < 0 || limiar > 255) {
        fprintf(stderr, "Erro: limiar deve ser um inteiro no intervalo [0, 255].\n");
        return 1;
    }

    // 2. ABERTURA E LEITURA DO ARQUIVO DE ENTRADA
    FILE *inputFile = fopen(input_filename, "r");
    if (!inputFile) {
        perror("Erro ao abrir o arquivo de entrada");
        return 1;
    }

    char magic[3];
    int width, height, max_val;
    if (fscanf(inputFile, "%2s", magic) != 1) {
        fprintf(stderr, "Erro lendo o cabeçalho do arquivo.\n");
        fclose(inputFile);
        return 1;
    }
    if (strcmp(magic, "P3") != 0) {
        fprintf(stderr, "Erro: O arquivo de entrada não é um PPM P3 válido.\n");
        fclose(inputFile);
        return 1;
    }

    int c;
    while ((c = fgetc(inputFile)) == ' ' || c == '\t' || c == '\n');
    if (c == '#') {
        while (fgetc(inputFile) != '\n');
    } else {
        ungetc(c, inputFile);
    }

    if (fscanf(inputFile, "%d %d %d", &width, &height, &max_val) != 3) {
        fprintf(stderr, "Erro lendo dimensões PPM.\n");
        fclose(inputFile);
        return 1;
    }

    printf("Lendo imagem '%s' (%d x %d)...\n", input_filename, width, height);
    printf("Parâmetros do filtro: M=%d, limiar=%d, sharpen_factor=%.2f\n", M, limiar, sharpen_factor);

    int n = width * height;

    Pixel *cpu_original_image = (Pixel *)malloc(n * sizeof(Pixel));
    Pixel *cpu_final_image = (Pixel *)malloc(n * sizeof(Pixel));

    for (int i = 0; i < width * height; i++) {
        fscanf(inputFile, "%hhu %hhu %hhu", &cpu_original_image[i].r, &cpu_original_image[i].g, &cpu_original_image[i].b);
    }    
    fclose(inputFile);

    // Device buffers
    Pixel *gpu_original_image, *gpu_final_image;
    unsigned char *gpu_grayscale_image, *gpu_blurred_image;

    Allocate_memmory(&gpu_original_image, &gpu_grayscale_image, &gpu_blurred_image, &gpu_final_image, width, height);

    hipMemcpy(gpu_original_image, cpu_original_image, n * sizeof(Pixel), hipMemcpyHostToDevice);

    int th_per_blk = 256;
    int blk_ct = (int)((n + th_per_blk - 1) / th_per_blk);

    gray_scale_transformation<<<blk_ct, th_per_blk>>>(gpu_original_image, gpu_grayscale_image, (int)n);
    hipDeviceSynchronize();
    printf("Fiz Gray Scale!\n");

    blur_transformation<<<blk_ct, th_per_blk>>>(gpu_original_image, gpu_grayscale_image, gpu_blurred_image, width, height, M);
    hipDeviceSynchronize();
    printf("Fiz Blur!\n");

    sharpen_kernel<<<blk_ct, th_per_blk>>>(gpu_original_image, gpu_blurred_image, gpu_final_image, (int)n, limiar, sharpen_factor);
    hipGetLastError();
    hipDeviceSynchronize();
    printf("Fiz Sharpen!\n");

    // Copy device -> host
    hipMemcpy(cpu_final_image, gpu_final_image, n * sizeof(Pixel), hipMemcpyDeviceToHost);

    printf("Escrevendo imagem de saída em '%s'...\n", output_filename);
    FILE *outputFile = fopen(output_filename, "w");
    if (!outputFile) {
        perror("Erro ao criar o arquivo de saída");
        Free_vectors(&gpu_original_image, &gpu_grayscale_image, &gpu_blurred_image, &gpu_final_image);
        free(cpu_original_image);
        free(cpu_final_image);
        return 1;
    }

    fprintf(outputFile, "P3\n%d %d\n%d\n", width, height, max_val);
    for (int i = 0; i < n; i++) {
        fprintf(outputFile, "%u %u %u\n", (unsigned int)cpu_final_image[i].r,
                (unsigned int)cpu_final_image[i].g, (unsigned int)cpu_final_image[i].b);
    }
    fclose(outputFile);

    Free_vectors(&gpu_original_image, &gpu_grayscale_image, &gpu_blurred_image, &gpu_final_image);
    free(cpu_original_image);
    free(cpu_final_image);

    printf("Filtro aplicado com sucesso!\n");
    return 0;
}
