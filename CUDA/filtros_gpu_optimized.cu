/* 
E.P 2 - Filtro de Realce Seletivo
 > Paralelização em CUDA

João Pedro Corrêa Silva	        				R.A: 11202321629
João Pedro Sousa Bianchim		    			R.A: 11201920729
Thiago Vinícius Pereira Graciano de Souza   	R.A: 11201722589

Professor: Emílio Francesquini
*/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

typedef struct {
    unsigned char r, g, b;
} Pixel;

// Função que Aloca Memória na GPU
void Allocate_memmory(Pixel **gpu_original_p, Pixel **gpu_blurred_p, Pixel **gpu_final_p, int width, int height) {
    int n = width * height;
    hipMalloc((void **)gpu_original_p, n * sizeof(Pixel));
    // gpu_blurred_p armazena a imagem borrada por canal (R,G,B) conforme especificação
    hipMalloc((void **)gpu_blurred_p, n * sizeof(Pixel));
    hipMalloc((void **)gpu_final_p, n * sizeof(Pixel));
}

// Função que Libera a memória alocada na GPU
void Free_vectors(Pixel **gpu_original_p, Pixel **gpu_blurred_p, Pixel **gpu_final_p) {
    hipFree(*gpu_original_p);
    hipFree(*gpu_blurred_p);
    hipFree(*gpu_final_p);
}

// Função que executa o cálculo para transformação em gray scale (cada thread processa um pixel)
__global__ void gray_scale_transformation(const Pixel *in, Pixel *out, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= n) return;
    unsigned char r = in[idx].r;
    unsigned char g = in[idx].g;
    unsigned char b = in[idx].b;
    unsigned char gray = (unsigned char)(0.299f * r + 0.587f * g + 0.114f * b);
    out[idx].r = gray;
    out[idx].g = gray;
    out[idx].b = gray;
}

// Função que executa o cálculo para transformação de blur (cada thread processa um pixel)
__global__ void blur_transformation(const Pixel *orig, Pixel *blur, int width, int height, int M) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int n = width * height;
    if (idx >= n) return;

    Pixel p = orig[idx];
    int radius = ((int)p.r + (int)p.g + (int)p.b) % M + 1;
    int x = idx % width;
    int y = idx / width;

    int x0 = max(0, x - radius);
    int x1 = min(width - 1, x + radius);
    int y0 = max(0, y - radius);
    int y1 = min(height - 1, y + radius);

    unsigned int sumR = 0, sumG = 0, sumB = 0;
    int count = 0;
    for (int yy = y0; yy <= y1; ++yy) {
        int base = yy * width;
        for (int xx = x0; xx <= x1; ++xx) {
            Pixel q = orig[base + xx];
            sumR += q.r;
            sumG += q.g;
            sumB += q.b;
            ++count;
        }
    }
    if (count == 0) count = 1;
    Pixel out;
    out.r = (unsigned char)(sumR / count);
    out.g = (unsigned char)(sumG / count);
    out.b = (unsigned char)(sumB / count);
    blur[idx] = out;
}

// Função para fazer o clamp e impedir valores diferentes dos permitidos para as cores
__device__ unsigned char clamp(int v) {
    if (v < 0) return 0;
    if (v > 255) return 255;
    return (unsigned char)v;
}

// Função que executa o cálculo para transformação de sharpen (cada thread processa um pixel)
__global__ void sharpen_kernel(const Pixel *orig, const Pixel *blur,
                               Pixel *out, int n, int limiar, float sharpen_factor) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = idx; i < n; i += stride) {
        Pixel p = orig[i];
        Pixel b = blur[i];

        if (p.r > limiar) {
            float new_r = (float)p.r + sharpen_factor * ((float)p.r - (float)b.r);
            float new_g = (float)p.g + sharpen_factor * ((float)p.g - (float)b.g);
            float new_b = (float)p.b + sharpen_factor * ((float)p.b - (float)b.b);

            out[i].r = clamp((int)new_r);
            out[i].g = clamp((int)new_g);
            out[i].b = clamp((int)new_b);
        } else {
            out[i] = p;
        }
    }
}

int main(int argc, char *argv[]) {
    if (argc != 6) {
        printf("Erro: Número incorreto de argumentos.\n");
        return 1;
    }

    const char *input_filename = argv[1];
    const char *output_filename = argv[2];
    int M = atoi(argv[3]);
    int limiar = atoi(argv[4]);
    float sharpen_factor = atof(argv[5]);

    // Abertura do arquivo
    if (M < 1) {
        fprintf(stderr, "Erro: M deve ser um inteiro maior ou igual a 1.\n");
        return 1;
    }
    if (limiar < 0 || limiar > 255) {
        fprintf(stderr, "Erro: limiar deve ser um inteiro no intervalo [0, 255].\n");
        return 1;
    }
    FILE *inputFile = fopen(input_filename, "r");
    if (!inputFile) {
        perror("Erro ao abrir o arquivo de entrada");
        return 1;
    }

    char magic[3];
    int width, height, max_val;
    if (fscanf(inputFile, "%2s\n%d %d\n%d\n", magic, &width, &height, &max_val) != 4) {
        fprintf(stderr, "Erro lendo dimensões PPM.\n");
        fclose(inputFile);
        return 1;
    }

    int n = width * height;

    // Criação das matrizes para armazenar a imagem na CPU
    Pixel *cpu_original_image = (Pixel *)malloc(n * sizeof(Pixel));
    Pixel *cpu_final_image = (Pixel *)malloc(n * sizeof(Pixel));

    // Leitura do arquivo
    for (int i = 0; i < width * height; i++) {
        fscanf(inputFile, "%hhu %hhu %hhu", &cpu_original_image[i].r, &cpu_original_image[i].g, &cpu_original_image[i].b);
    }    
    fclose(inputFile);

    // Cálculos na GPU
    Pixel *gpu_original_image, *gpu_blurred_image, *gpu_final_image;

    Allocate_memmory(&gpu_original_image, &gpu_blurred_image, &gpu_final_image, width, height);

    hipMemcpy(gpu_original_image, cpu_original_image, n * sizeof(Pixel), hipMemcpyHostToDevice);

    int th_per_blk = 256;
    int blk_ct = (int)((n + th_per_blk - 1) / th_per_blk);

    // Esteira de Aplicação dos Filtros
    blur_transformation<<<blk_ct, th_per_blk>>>(gpu_original_image, gpu_blurred_image, width, height, M);
    hipDeviceSynchronize();

    sharpen_kernel<<<blk_ct, th_per_blk>>>(gpu_original_image, gpu_blurred_image, gpu_final_image, (int)n, limiar, sharpen_factor);
    hipDeviceSynchronize();

    // Aplica escala de cinza como última transformação
    gray_scale_transformation<<<blk_ct, th_per_blk>>>(gpu_final_image, gpu_final_image, (int)n);
    hipDeviceSynchronize();

    hipMemcpy(cpu_final_image, gpu_final_image, n * sizeof(Pixel), hipMemcpyDeviceToHost);

    //Escrita do Arquivo Final
    FILE *outputFile = fopen(output_filename, "w");
    if (!outputFile) {
        perror("Erro ao criar o arquivo de saída");
        //Liberação de Memória
        Free_vectors(&gpu_original_image, &gpu_blurred_image, &gpu_final_image);
        free(cpu_original_image);
        free(cpu_final_image);
        return 1;
    }

    fprintf(outputFile, "P3\n%d %d\n%d\n", width, height, max_val);
    for (int i = 0; i < n; i++) {
        fprintf(outputFile, "%u %u %u\n", (unsigned int)cpu_final_image[i].r,
                (unsigned int)cpu_final_image[i].g, (unsigned int)cpu_final_image[i].b);
    }
    fclose(outputFile);

    //Liberação de Memória
    Free_vectors(&gpu_original_image, &gpu_blurred_image, &gpu_final_image);
    free(cpu_original_image);
    free(cpu_final_image);

    return 0;
}
